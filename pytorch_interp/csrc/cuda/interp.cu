#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace extension_interp {

__global__ void bilinear_interpolation_kernel_CUDA(
  float * G, float * F, 
  float * xpts, float * ypts, 
  const int M1, const int M2, const int N,
  const float dx, const float dy, 
  const float * x, const float * y) {

  int k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k < N) {
    const int    ind_x  = floor((xpts[k]-x[0])/dx);
    const int    ind_xp = ind_x+1;

    const int    ind_y  = floor((ypts[k]-y[0])/dy);
    const int    ind_yp = ind_y+1;

    if ( 0 <= ind_x && ind_xp  < M1 && 0 <= ind_y && ind_yp  < M2 ) {
      const float w11 = (x[ind_xp]-xpts[k])*(y[ind_yp]-ypts[k]);
      const float w12 = (x[ind_xp]-xpts[k])*(ypts[k]-y[ind_y]);
      const float w21 = (xpts[k]-x[ind_x])*(y[ind_yp]-ypts[k]);
      const float w22 = (xpts[k]-x[ind_x])*(ypts[k]-y[ind_y]);
      const float numerator = w11*F[ind_x*M2+ind_y] + w12*F[ind_x*M2+ind_y+1] + w21*F[(ind_x+1)*M2+ind_y] + w22*F[(ind_x+1)*M2+ind_y+1];
      const float denominator = (x[ind_xp]-x[ind_x])*(y[ind_yp]-y[ind_y]);
      G[k] = numerator/denominator;
    }
    else{
      G[k] = 0.f;
    }

  }
}

void bilinear_interp_cuda(
    const at::Tensor& F, 
    at::Tensor& G, 
    const at::Tensor& x,
    const at::Tensor& y,
    const at::Tensor& xpt,
    const at::Tensor& ypt,
    const int64_t M1, 
    const int64_t M2,
    const double dx, 
    const double dy
  ) {

  at::Tensor F_contig = F.contiguous();
  float* F_ptr = F_contig.data_ptr<float>();

  at::Tensor xpt_contig = xpt.contiguous();
  float* xpt_ptr = xpt_contig.data_ptr<float>();

  at::Tensor ypt_contig = ypt.contiguous();
  float* ypt_ptr = ypt_contig.data_ptr<float>();

  at::Tensor x_contig = x.contiguous();
  const float* x_ptr = x_contig.data_ptr<float>();

  at::Tensor y_contig = y.contiguous();
  const float* y_ptr = y_contig.data_ptr<float>();

  const int N  = xpt_contig.numel();

  float* G_ptr = G.data_ptr<float>();

  bilinear_interpolation_kernel_CUDA<<<(N+255)/256, 256>>>(
                                    G_ptr, F_ptr,
                                    xpt_ptr, ypt_ptr,
                                    M1, M2, N,
                                    dx, dy,
                                    x_ptr, y_ptr);
}

// Registers CUDA implementation
TORCH_LIBRARY_IMPL(extension_interp, CUDA, m) {
  m.impl("bilinear_interp", &bilinear_interp_cuda);
}

}
